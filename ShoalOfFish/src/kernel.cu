#include "hip/hip_runtime.h"
#include <cstdio>
#include "kernel.cuh"
#include "Fish.h"
#include <thrust>

#define BLOCKS 256
#define THREADS 1024

Fish *dev_fishes;
Fish *dev_gathered_fishes;
int *dev_indices;
int *dev_grid_first;
int *dev_grid_last;
int *dev_cell_idx;
int N;

// global
void assign_grid(Fish *fishes, float cell_size, int* cell_idx, int* indices)
{
    int grid_size = (int)(2.0f / cell_size) + 1;
    for (int i = 0; i < N; i++)
    {
        float x = fishes[i].x + 1.0f;
        float y = fishes[i].y + 1.0f;
        int r = (int)(y / cell_size);
        int c = (int)(x / cell_size);
        dev_cell_idx[i] = r * grid_size + c;
        indices[i] = i;
    }
}

void make_calculations_cuda(Fish *fishes, float r1, float r2, float turnCoef, float cohensionCoef, float avoidCoef, float alignCoef, float predatorsCoef,
    float preyCoef, float maxV, float minV, float curX, float curY, float curActive, bool predatorMode)
{
    hipError_t cudaStatus;
    
    float cell_size = r2 * 2;
    int grid_size = (int)(2.0f / cell_size) + 1;
    int grid_length = (grid_size) * (grid_size);

    cudaStatus = hipMemcpy(dev_fishes, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemset(dev_grid_first, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }
    cudaStatus = hipMemset(dev_grid_last, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }

    assign_grid(dev_fishes, cell_size, dev_cell_idx, dev_indices);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
    }

    auto fishes_pointer = thrust::device_pointer_cast(dev_fishes);
    auto gathered_fishes_pointer = thrust::device_pointer_cast(dev_gathered_fishes);
    auto cell_idx_pointer = thrust::device_pointer_cast(dev_cell_idx);
    auto indices_pointer = thrust::device_pointer_cast(dev_indices);


}

void init_cuda(int n, int grid_length)
{
    hipError_t cudaStatus;

    N = n;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    cudaStatus = hipMalloc((void**)&dev_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_gathered_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_indices, N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_first, grid_length * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_last, grid_length * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_cell_idx, N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
}

void free_cuda()
{
    hipFree(dev_fishes);
    hipFree(dev_gathered_fishes);
    hipFree(dev_indices);
    hipFree(dev_grid_first);
    hipFree(dev_grid_last);
    hipFree(dev_cell_idx);
}