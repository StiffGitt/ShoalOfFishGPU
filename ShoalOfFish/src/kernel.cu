#include "hip/hip_runtime.h"
#include <cstdio>
#include "kernel.cuh"
#include "Fish.h"
#include <thrust>

#define BLOCKS 256
#define THREADS 1024

Fish *dev_fishes;
Fish *dev_gathered_fishes;
int *dev_indices;
int *dev_grid_first;
int *dev_grid_last;
int *dev_cell_idx;
int N;

// global
void assign_grid(Fish *fishes, float cell_size, int* cell_idx, int* indices)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int grid_size = (int)(2.0f / cell_size) + 1;
    float x = fishes[idx].x + 1.0f;
    float y = fishes[idx].y + 1.0f;
    int r = (int)(y / cell_size);
    int c = (int)(x / cell_size);
    dev_cell_idx[idx] = r * grid_size + c;
    indices[idx] = idx;
}

// global
void find_border_cells(int* grid_first, int* grid_last, int* cell_idx)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx == 0)
        grid_first[cell_idx[0]] = 0;
    int cur_cell = cell_idx[idx];
    int prev_cell = cell_idx[idx - 1];
    if (cur_cell != prev_cell)
    {
        grid_last[prev_cell] = idx;
        grid_first[cur_cell] = idx;
    }
    if (idx == N - 1)
        grid_last[cur_cell] = N;
}

void make_calculations_cuda(Fish *fishes, float r1, float r2, float turnCoef, float cohensionCoef, float avoidCoef, float alignCoef, float predatorsCoef,
    float preyCoef, float maxV, float minV, float curX, float curY, float curActive, bool predatorMode)
{
    hipError_t cudaStatus;
    
    float cell_size = r2 * 2;
    int grid_size = (int)(2.0f / cell_size) + 1;
    int grid_length = (grid_size) * (grid_size);

    cudaStatus = hipMemcpy(dev_fishes, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemset(dev_grid_first, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }
    cudaStatus = hipMemset(dev_grid_last, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }

    assign_grid(dev_fishes, cell_size, dev_cell_idx, dev_indices);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
    }

    auto fishes_pointer = thrust::device_pointer_cast(dev_fishes);
    auto gathered_fishes_pointer = thrust::device_pointer_cast(dev_gathered_fishes);
    auto cell_idx_pointer = thrust::device_pointer_cast(dev_cell_idx);
    auto indices_pointer = thrust::device_pointer_cast(dev_indices);

    thrust::sort_by_key(cell_idx_pointer, cell_idx_pointer + N, indices_pointer);

}

void init_cuda(int n, int grid_length)
{
    hipError_t cudaStatus;

    N = n;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    cudaStatus = hipMalloc((void**)&dev_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_gathered_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_indices, N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_first, grid_length * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_last, grid_length * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_cell_idx, N * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
}

void free_cuda()
{
    hipFree(dev_fishes);
    hipFree(dev_gathered_fishes);
    hipFree(dev_indices);
    hipFree(dev_grid_first);
    hipFree(dev_grid_last);
    hipFree(dev_cell_idx);
}