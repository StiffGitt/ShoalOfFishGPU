#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include "kernel.cuh"
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include "Consts.h"

#define BLOCKS 256
#define THREADS 1024

Fish *dev_fishes;
Fish *dev_gathered_fishes;
int *dev_indices;
int *dev_grid_first;
int *dev_grid_last;
int *dev_cell_idx;
float *dev_v_x;
float *dev_v_y;
float *dev_vertices;

__global__ void assign_grid(Fish *fishes, float cell_size, int* cell_idx, int* indices)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int grid_size = (int)(2.0f / cell_size) + 1;
    float x = fishes[idx].x + 1.0f;
    float y = fishes[idx].y + 1.0f;
    int r = (int)(y / cell_size);
    int c = (int)(x / cell_size);
    cell_idx[idx] = r * grid_size + c;
    indices[idx] = idx;
}

__global__ void find_border_cells(int* grid_first, int* grid_last, int* cell_idx)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    if (idx == 0)
    {
        grid_first[cell_idx[0]] = 0;
        return;
    }
    int cur_cell = cell_idx[idx];
    int prev_cell = cell_idx[idx - 1];
    if (cur_cell != prev_cell)
    {
        grid_last[prev_cell] = idx;
        grid_first[cur_cell] = idx;
    }
    if (idx == N - 1)
        grid_last[cur_cell] = N;
}

__global__ void calculate_v(Fish* gathered_fishes, int* grid_first, int* grid_last, int* cell_idx, float* v_x, float* v_y, float r1, float r2, int grid_size, float cohensionCoef, float avoidCoef, float alignCoef, float predatorsCoef, float preyCoef, bool predatorMode)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int grid_length = (grid_size) * (grid_size);
    float r1sq = r1 * r1;

    float x = gathered_fishes[idx].x;
    float y = gathered_fishes[idx].y;
    float vx = gathered_fishes[idx].dx;
    float vy = gathered_fishes[idx].dy;
    float cumX = 0.0, cumY = 0.0, cumVx = 0.0, cumVy = 0.0, visibleFriendlyCount = 0.0, visiblePreyCount = 0.0,
        closestPredatorX = -1.0, closestPredatorY = -1.0f, closeDx = 0.0, closeDy = 0.0, cumXP = 0.0, cumYP = 0.0,
        closestPredatorDsq = 8.0f;
    int cell = cell_idx[idx];
    int cells_to_check[] = { cell - 1, cell, cell + 1,
        cell - grid_size - 1, cell - grid_size, cell - grid_size + 1,
        cell - grid_size + 1, cell + grid_size, cell + grid_size + 1 };
    
    for(int cidx = 0; cidx < 9; cidx++)
    {
        int nc = cells_to_check[cidx];
        if (nc < 0 || nc > grid_length || grid_first[nc] < 0)
            continue;
        for (int j = grid_first[nc]; j < grid_last[nc]; j++)
        {
            if (j == idx)
                continue;
            float xj = gathered_fishes[j].x;
            float yj = gathered_fishes[j].y;
            float dx = x - xj;
            float dy = y - yj;

            if (fabsf(dx) < r2 && fabsf(dy) < r2)
            {
                float dsq = dx * dx + dy * dy;
                if (dsq < r2)
                {
                    // Avoid predators
                    if (gathered_fishes[idx].species < gathered_fishes[j].species)
                    {
                        if (closestPredatorDsq > dsq)
                        {
                            closestPredatorDsq = dsq;
                            closestPredatorX = xj;
                            closestPredatorY = yj;
                        }
                    }
                    // Hunt prey
                    if (gathered_fishes[idx].species > gathered_fishes[j].species)
                    {
                        visiblePreyCount++;
                        cumXP += xj;
                        cumYP += yj;
                    }
                    if (dsq < r1sq)
                    {
                        // Separation
                        closeDx += (x - xj); /** (1 - (dx / r1));*/
                        closeDy += (y - yj); /** (1 - (dy / r1));*/
                    }
                    else
                    {
                        if (gathered_fishes[idx].species == gathered_fishes[j].species && gathered_fishes[idx].species <= 1)
                        {
                            visibleFriendlyCount++;
                            // Alignment
                            cumVx += gathered_fishes[j].dx;
                            cumVy += gathered_fishes[j].dy;

                            // Cohension
                            cumX += xj;
                            cumY += yj;
                        }
                    }
                }
            }
        }
    }
    
    // Avoid predators
    if (predatorMode && closestPredatorDsq < r2)
    {
        vx += (x - closestPredatorX) * predatorsCoef;
        vy += (y - closestPredatorY) * predatorsCoef;
    }

    //// Chase prey
    if (predatorMode && visiblePreyCount > 0)
    {
        vx += ((cumXP / visiblePreyCount) - x) * preyCoef;
        vy += ((cumYP / visiblePreyCount) - y) * preyCoef;
    }

    //// Separation
    vx += closeDx * avoidCoef;
    vy += closeDy * avoidCoef;

    if (visibleFriendlyCount > 0)
    {
        // Alignment
        vx += ((cumVx / visibleFriendlyCount) - gathered_fishes[idx].dx) * alignCoef;
        vy += ((cumVy / visibleFriendlyCount) - gathered_fishes[idx].dy) * alignCoef;

        // Cohension
        vx += ((cumX / visibleFriendlyCount) - x) * cohensionCoef;
        vy += ((cumY / visibleFriendlyCount) - y) * cohensionCoef;
    }

    v_x[idx] = vx;
    v_y[idx] = vy;
}

__global__ void scale_and_move(Fish* gathered_fishes, float* v_x, float* v_y, float maxV, float minV, float curX, float curY, float curActive)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    float x = gathered_fishes[idx].x;
    float y = gathered_fishes[idx].y;
    float vx = v_x[idx];
    float vy = v_y[idx];

    // Turn from edges
    bool isTurning = false;
    if (x < LEFT_EDGE && vx < minV)
    {
        isTurning = true;
        if (x < -1.0 + MARGIN && vx < 0)
            vx = -vx;
        else
            vx += TURN_COEF + (vx * vx) / (x + 1.0f);
    }
    if (x > RIGHT_EDGE && vx > -minV)
    {
        isTurning = true;
        if (x > 1.0 - MARGIN && vx > 0)
            vx = -vx;
        else
            vx -= TURN_COEF + (vx * vx) / (1.0f - x);
    }

    if (y < BOTTOM_EDGE && vy < minV)
    {
        isTurning = true;
        if (y < -1.0 + MARGIN && vy < 0)
            vy = -vy;
        else
            vy += TURN_COEF + (vy * vy) / (y + 1.0f);
    }
    if (y > TOP_EDGE && vy > -minV)
    {
        isTurning = true;
        if (y > 1.0 - MARGIN && vy > 0)
            vy = -vy;
        else
            vy -= TURN_COEF + (vy * vy) / (1.0f - y);
    }

    float dcx = x - curX;
    float dcy = y - curY;
    if (curActive && dcx * dcx + dcy * dcy < CURSOR_RANGE * CURSOR_RANGE)
    {
        vx += dcx * CURSOR_COEF;
        vy += dcy * CURSOR_COEF;
    }

    //// Adjust velocity to minmax
    float v = sqrtf(vx * vx + vy * vy);
    if (v < minV && !isTurning)
    {
        vx = (vx / v) * minV;
        vy = (vy / v) * minV;
    }
    else if (v > maxV)
    {
        vx = (vx / v) * maxV;
        vy = (vy / v) * maxV;
    }

    gathered_fishes[idx].dx = vx;
    gathered_fishes[idx].dy = vy;

    gathered_fishes[idx].x += vx;
    gathered_fishes[idx].y += vy;
}

__global__ void get_vertices(Fish *fishes, float *vertices)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    float A[3] = { 0.005f, 0.007f, 0.01f };
    float H[3] = { 0.02f, 0.03f, 0.04f };

    float x = fishes[idx].x;
    float y = fishes[idx].y;
    float dx = fishes[idx].dx;
    float dy = fishes[idx].dy;
    int species = fishes[idx].species;
    float d = sqrtf(dx * dx + dy * dy);

    vertices[idx * 3 * ATTR_COUNT] = x - A[species] * (dy / d);
    vertices[idx * 3 * ATTR_COUNT + 1] = y + A[species] * (dx / d);
    vertices[idx * 3 * ATTR_COUNT + 2] = GET_R(species);
    vertices[idx * 3 * ATTR_COUNT + 3] = GET_G(species);
    vertices[idx * 3 * ATTR_COUNT + 4] = GET_B(species);
    
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT] = x + A[species] * (dy / d);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT + 1] = y - A[species] * (dx / d);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT + 2] = GET_R(species);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT + 3] = GET_G(species);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT + 4] = GET_B(species);
    
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT * 2] = x + H[species] * (dx / d);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT * 2 + 1] = y + H[species] * (dy / d);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT * 2 + 2] = GET_R(species);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT * 2 + 3] = GET_G(species);
    vertices[idx * 3 * ATTR_COUNT + ATTR_COUNT * 2 + 4] = GET_B(species);
}

void make_calculations_cuda(float *vertices, float r1, float r2, float turnCoef, float cohensionCoef, float avoidCoef, float alignCoef, float predatorsCoef,
    float preyCoef, float maxV, float minV, float curX, float curY, float curActive, bool predatorMode)
{
    hipError_t cudaStatus;
    
    float cell_size = r2 * 2;
    int grid_size = (int)(2.0f / cell_size) + 1;
    int grid_length = (grid_size) * (grid_size);

    cudaStatus = hipMemset(dev_grid_first, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }
    cudaStatus = hipMemset(dev_grid_last, -1, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
    }

    assign_grid << < BLOCKS, THREADS >> > (dev_fishes, cell_size, dev_cell_idx, dev_indices);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
    }
    
    auto cell_idx_pointer = thrust::device_pointer_cast(dev_cell_idx);
    auto indices_pointer = thrust::device_pointer_cast(dev_indices);
    thrust::sort_by_key(cell_idx_pointer, cell_idx_pointer + N, indices_pointer);

    find_border_cells << < BLOCKS, THREADS >> > (dev_grid_first, dev_grid_last, dev_cell_idx);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
    }

    auto fishes_pointer = thrust::device_pointer_cast(dev_fishes);
    auto gathered_fishes_pointer = thrust::device_pointer_cast(dev_gathered_fishes);
    thrust::gather(indices_pointer, indices_pointer + N, fishes_pointer, gathered_fishes_pointer);

    calculate_v << < BLOCKS, THREADS >> > (dev_gathered_fishes, dev_grid_first, dev_grid_last, dev_cell_idx, dev_v_x, dev_v_y, r1, r2, grid_size, cohensionCoef, avoidCoef, alignCoef, predatorsCoef, preyCoef, predatorMode);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
    }

    scale_and_move << < BLOCKS, THREADS >> > (dev_gathered_fishes, dev_v_x, dev_v_y, maxV, minV, curX, curY, curActive);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scale_and_move_cudaDeviceSynchronize failed!");
    }

    get_vertices << < BLOCKS, THREADS >> > (dev_gathered_fishes, dev_vertices);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scale_and_move_cudaDeviceSynchronize failed!");
    }

    cudaStatus = hipMemcpy(vertices, dev_vertices, (N * 3 * ATTR_COUNT) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vertices hipMemcpy failed!");
    }

    // Swap fishes arrays
    Fish* temp = dev_fishes;
    dev_fishes = dev_gathered_fishes;
    dev_gathered_fishes = temp;
}

void init_cuda(int grid_length, Fish *fishes)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }
    cudaStatus = hipMalloc((void**)&dev_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_gathered_fishes, N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_indices, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_first, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_grid_last, grid_length * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_cell_idx, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_v_x, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_v_y, N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc((void**)&dev_vertices, (N * 3 * ATTR_COUNT) * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMemcpy(dev_fishes, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
}

void free_cuda()
{
    hipFree(dev_fishes);
    hipFree(dev_gathered_fishes);
    hipFree(dev_indices);
    hipFree(dev_grid_first);
    hipFree(dev_grid_last);
    hipFree(dev_cell_idx);
    hipFree(dev_v_x);
    hipFree(dev_v_y);
    hipFree(dev_vertices);
}